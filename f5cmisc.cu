#include "f5c.cuh"
#include <stdint.h>

void gpu_assert(const char* file, uint64_t line) {
    hipError_t code = hipGetLastError();
    if (code != hipSuccess) {
        fprintf(stderr, "Cuda error: %s \n in file : %s line number : %d\n",
                hipGetErrorString(code), file, line);
        exit(-1);
    }
}

int32_t cuda_exists() {
    //check cuda devices
    int32_t nDevices;
    hipGetDeviceCount(&nDevices);
    if (nDevices == 0) {
        fprintf(stderr, "No CUDA device found. Use the CPU version\n");
        exit(1);
    }

    return nDevices;
}

uint64_t cuda_freemem(int32_t devicenum) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devicenum);
    fprintf(stderr, "Device name: %s\n", prop.name);
    uint64_t golabalmem = prop.totalGlobalMem;
    fprintf(stderr, "Total global memory: %lf GB\n",
            (golabalmem / double(1024 * 1024 * 1024)));
    uint64_t freemem, total;
    hipMemGetInfo(&freemem, &total);
    fprintf(stderr, "%lf GB free of total %lf GB\n",
            freemem / double(1024 * 1024 * 1024),
            total / double(1024 * 1024 * 1024));

    return freemem;
}
